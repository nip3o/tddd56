#include "hip/hip_runtime.h"
// Matrix addition, GPU version

#include <stdio.h>

const int N = 16;  // matrix size

const int GRIDSIZE = 1;
const int BLOCKSIZE = 16;


__global__
void multiply(float *a, float *b, float *c) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    c[x*sizeof(float) + y] = 76;//a[index] + b[index];
}

// void add_matrix(float *a, float *b, float *c, int N)
// {
//     int index;
    
//     for (int i = 0; i < N; i++) {
//         for (int j = 0; j < N; j++) {
//             index = i + j*N;
//             c[index] = a[index] + b[index];
//         }
//     }
// }

int main()
{
    const int size = N*N*sizeof(float);

    float *a = new float[N*N];
    float *b = new float[N*N];
    float *c = new float[N*N];

    float *a_gpu, *b_gpu, *c_gpu;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i+j*N] = 10 + i;
            b[i+j*N] = (float)j / N;
        }
    }

    hipMalloc((void**)&a_gpu, size);
    hipMalloc((void**)&b_gpu, size);
    hipMalloc((void**)&c_gpu, size);

    hipMemcpy( a_gpu, a, size, hipMemcpyHostToDevice ); 
    hipMemcpy( b_gpu, b, size, hipMemcpyHostToDevice ); 

    dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );
    dim3 dimGrid( GRIDSIZE, GRIDSIZE );

    multiply<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu);
    hipDeviceSynchronize();

    hipMemcpy( c, c_gpu, size, hipMemcpyDeviceToHost ); 

    //add_matrix(a, b, c, N);

    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%0.2f ", c[i+j*N]);
        }
        printf("\n");
    }
}
