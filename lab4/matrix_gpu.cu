#include "hip/hip_runtime.h"
// Matrix addition, GPU version
// nvcc matrix_gpu.cu -L /usr/local/cuda/lib -lcudart -arch=sm_20 -o matrix_gpu 

#include <stdio.h>

const int BLOCKSIZE = 32;
const int GRIDSIZE = 1;


__global__
void multiply(float *a, float *b, float *c, int N) {
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    float sum = 0;

    sum = a[row*N + col] + b[row*N + col];
    c[N*row + col] = sum;
}

int main()
{
    const int N = 512;  // matrix size
    const int size = N*N*sizeof(float);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *a = new float[N*N];
    float *b = new float[N*N];
    float *c = new float[N*N];

    float *a_gpu, *b_gpu, *c_gpu;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i+j*N] = 10 + i;
            b[i+j*N] = (float)j / N;
        }
    }

    hipMalloc((void**)&a_gpu, size);
    hipMalloc((void**)&b_gpu, size);
    hipMalloc((void**)&c_gpu, size);

    hipMemcpy( a_gpu, a, size, hipMemcpyHostToDevice ); 
    hipMemcpy( b_gpu, b, size, hipMemcpyHostToDevice ); 

    dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );
    dim3 dimGrid( GRIDSIZE, GRIDSIZE );

    
    hipEventRecord(start, 0);


    multiply<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu, N);
    hipDeviceSynchronize();

    hipMemcpy( c, c_gpu, size, hipMemcpyDeviceToHost ); 


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float theTime;
    hipEventElapsedTime(&theTime, start, stop);

    printf("Things took %f ms\n", theTime);


    // for (int i = 0; i < N; i++) {
    //     for (int j = 0; j < N; j++) {
    //         printf("%0.2f ", c[i+j*N]);
    //     }
    //     printf("\n");
    // }
}
